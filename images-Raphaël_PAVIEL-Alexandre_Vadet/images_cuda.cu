#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <iostream>
#define RGBSIZE 3
using namespace cv;


__global__ void noiretblanc( unsigned char * in, unsigned char * out, std::size_t colonnes, std::size_t lignes ) {
  auto colonne = blockIdx.x * blockDim.x + threadIdx.x;
  auto ligne = blockIdx.y * blockDim.y + threadIdx.y;

  if( colonne < colonnes && ligne < lignes ) {
    int pos = ligne*colonnes+colonne;
    int posG = pos*RGBSIZE;

    unsigned char r = in[posG];
    unsigned char b = in[posG+1];
    unsigned char g = in[posG+2];

    out[posG]=0.21f*r+0.71f*g+0.07f*b;
    out[posG+1]=0.21f*r+0.71f*g+0.07f*b;
    out[posG+2]=0.21f*r+0.71f*g+0.07f*b;
  }
}


__global__ void retourner( unsigned char * in, unsigned char * out, std::size_t colonnes, std::size_t lignes ) {
  auto colonne = blockIdx.x * blockDim.x + threadIdx.x;
  auto ligne = blockIdx.y * blockDim.y + threadIdx.y;

  if( colonne < colonnes && ligne < lignes ) {

    int pos = RGBSIZE * ( ligne * colonnes + colonne );
    int oppose = colonnes*lignes*RGBSIZE - pos;

    auto rbis = in[oppose];
    auto gbis = in[oppose + 1];
    auto bbis = in[oppose + 2];

    out[pos] = rbis;
    out[pos + 1] = gbis;
    out[pos + 2] = bbis;
  }
}

__global__ void detectionContours(unsigned char * in, unsigned char * out, std::size_t colonnes, std::size_t lignes) {
  auto colonne = blockIdx.x * blockDim.x + threadIdx.x;
  auto ligne = blockIdx.y * blockDim.y + threadIdx.y;

  if (ligne >= 1 && ligne < lignes - 1 && colonne >= 1 && colonne < colonnes - 1)
  {
      for (std::size_t i = 0; i < RGBSIZE; ++i)
      {
          unsigned char p_h = in[RGBSIZE * ((ligne - 1) * colonnes + colonne) + i];
          unsigned char p_g = in[RGBSIZE * (ligne * colonnes + colonne - 1) + i];
          unsigned char pixel = in[RGBSIZE * (ligne * colonnes + colonne) + i];
          unsigned char p_d = in[RGBSIZE * (ligne * colonnes + colonne + 1) + i];
          unsigned char p_b = in[RGBSIZE * ((ligne + 1) * colonnes + colonne) + i];

          int resultat = p_h + p_g + (-4*pixel) + p_d + p_b ;
          if (resultat > 255)
          {
            resultat = 255;
          }
          if (resultat < 0)
          {
            resultat = 0;
          }
          out[RGBSIZE * (ligne * colonnes + colonne) + i] = resultat;
      }
  }
}

__global__ void ameliorationNettete(unsigned char * in, unsigned char * out, std::size_t colonnes, std::size_t lignes) {
  auto colonne = blockIdx.x * blockDim.x + threadIdx.x;
  auto ligne = blockIdx.y * blockDim.y + threadIdx.y;

  if (ligne >= 1 && ligne < lignes - 1 && colonne >= 1 && colonne < colonnes - 1)
  {
      for (std::size_t i = 0; i < RGBSIZE; ++i)
      {
          unsigned char p_h = in[RGBSIZE * ((ligne - 1) * colonnes + colonne) + i];
          unsigned char p_g = in[RGBSIZE * (ligne * colonnes + colonne - 1) + i];
          unsigned char pixel = in[RGBSIZE * (ligne * colonnes + colonne) + i];
          unsigned char p_d = in[RGBSIZE * (ligne * colonnes + colonne + 1) + i];
          unsigned char p_b = in[RGBSIZE * ((ligne + 1) * colonnes + colonne) + i];

          int resultat = -p_h - p_g + (5*pixel) - p_d - p_b ;
          if (resultat > 255)
          {
            resultat = 255;
          }
          if (resultat < 0)
          {
            resultat = 0;
          }
          out[RGBSIZE * (ligne * colonnes + colonne) + i] = resultat;
      }
  }
}

__global__ void flou(unsigned char * in, unsigned char * out, std::size_t colonnes, std::size_t lignes) {
  auto colonne = blockIdx.x * blockDim.x + threadIdx.x;
  auto ligne = blockIdx.y * blockDim.y + threadIdx.y;

  if (ligne >= 1 && ligne < lignes - 1 && colonne >= 1 && colonne < colonnes - 1)
  {
      for (std::size_t i = 0; i < RGBSIZE; ++i)
      {
          unsigned char p_hg = in[RGBSIZE * ((ligne - 1) * colonnes + colonne - 1) + i];
          unsigned char p_h = in[RGBSIZE * ((ligne - 1) * colonnes + colonne) + i];
          unsigned char p_hd = in[RGBSIZE * ((ligne - 1) * colonnes + colonne + 1) + i];
          unsigned char p_g = in[RGBSIZE * (ligne * colonnes + colonne - 1) + i];
          unsigned char pixel = in[RGBSIZE * (ligne * colonnes + colonne) + i];
          unsigned char p_d = in[RGBSIZE * (ligne * colonnes + colonne + 1) + i];
          unsigned char p_bg = in[RGBSIZE * ((ligne + 1) * colonnes + colonne - 1) + i];
          unsigned char p_b = in[RGBSIZE * ((ligne + 1) * colonnes + colonne) + i];
          unsigned char p_bd = in[RGBSIZE * ((ligne + 1) * colonnes + colonne + 1) + i];

          int resultat = (p_hg + p_h + p_hd + p_g + pixel + p_d + p_bg + p_b + p_bd)/9;
          if (resultat > 255)
          {
            resultat = 255;
          }
          if (resultat < 0)
          {
            resultat = 0;
          }
          out[RGBSIZE * (ligne * colonnes + colonne) + i] = resultat;
      }
  }
}

int main()
{
  cv::Mat m_in = cv::imread("image.jpeg", cv::IMREAD_UNCHANGED );
  cv::Mat m_out = m_in;

  auto lignes = m_in.rows;
  auto colonnes = m_in.cols;

  unsigned char * matrice_out;
  unsigned char * matrice_in;
  hipMalloc( &matrice_in, RGBSIZE*lignes * colonnes );
  hipMalloc( &matrice_out, RGBSIZE*lignes * colonnes );

  hipMemcpy( matrice_in, m_in.data, RGBSIZE * lignes * colonnes, hipMemcpyHostToDevice );

  dim3 t( 32, 32 );
  dim3 b( ( colonnes - 1) / t.x + 1 , ( lignes - 1 ) / t.y + 1 );

  int i;
  std::cout << "Entrez le filtre que vous voulez appliquer (1: Noir et Blanc // 2: Retourner // 3: Detection contours // 4: Amélioration de la netteté // 5: Flouter) : ";
  std::cin >> i;

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  if( i == 1 )
  {
    noiretblanc<<< b, t >>>(matrice_in, matrice_out, colonnes, lignes);

    hipMemcpy( m_out.data, matrice_out,RGBSIZE*lignes * colonnes, hipMemcpyDeviceToHost );

    cv::imwrite( "./resultat/NoirEtBlanc_CUDA.jpeg", m_out );

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float tempsexecution;
    hipEventElapsedTime(&tempsexecution, start, stop);
    std::cout << "Temps_NoirEtBlanc: " << tempsexecution << " millisecondes" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }
  else if( i == 2 )
  {
    retourner<<< b, t >>>(matrice_in, matrice_out, colonnes, lignes);

    hipMemcpy( m_out.data, matrice_out,RGBSIZE*lignes * colonnes, hipMemcpyDeviceToHost );

    cv::imwrite( "./resultat/Retourner_CUDA.jpeg", m_out );
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float tempsexecution;
    hipEventElapsedTime(&tempsexecution, start, stop);
    std::cout << "Temps_Retourner: " << tempsexecution << " millisecondes" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }
  else if( i == 3 )
  {
    detectionContours<<< b, t >>>(matrice_in, matrice_out, colonnes, lignes);

    hipMemcpy( m_out.data, matrice_out,RGBSIZE*lignes * colonnes, hipMemcpyDeviceToHost );

    cv::imwrite( "./resultat/DetectionContours_CUDA.jpeg", m_out );
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float tempsexecution;
    hipEventElapsedTime(&tempsexecution, start, stop);
    std::cout << "Temps_DetectionContours: " << tempsexecution << " millisecondes" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }
  else if( i == 4 )
  {
    ameliorationNettete<<< b, t >>>(matrice_in, matrice_out, colonnes, lignes);

    hipMemcpy( m_out.data, matrice_out,RGBSIZE*lignes * colonnes, hipMemcpyDeviceToHost );

    cv::imwrite( "./resultat/AmeliorationNettete_CUDA.jpeg", m_out );
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float tempsexecution;
    hipEventElapsedTime(&tempsexecution, start, stop);
    std::cout << "Temps_AmeliorationNettete: " << tempsexecution << " millisecondes" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }
  else if( i == 5 )
  {
    flou<<< b, t >>>(matrice_in, matrice_out, colonnes, lignes);

    hipMemcpy( m_out.data, matrice_out,RGBSIZE*lignes * colonnes, hipMemcpyDeviceToHost );

    cv::imwrite( "./resultat/Flouter_CUDA.jpeg", m_out );
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float tempsexecution;
    hipEventElapsedTime(&tempsexecution, start, stop);
    std::cout << "Temps_Flouter: " << tempsexecution << " millisecondes" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }
  else
  {
    std::cout << "Opération impossible" << std::endl;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }

  hipFree( matrice_in);
  hipFree( matrice_out);

  return 0;
}
